#include "hip/hip_runtime.h"
#include "Filter.cuh"

#ifdef USE_CUDA

#include "Filter.hpp"
#include "Image.hpp"
#include "Pixel.hpp"
#include "Exception.hpp"

#include <iostream>

#define CUDA_BLOCK_COUNT 128
#define CUDA_THREAD_COUNT 128

__global__ 
void ApplyFilterCuda(void* kernel, uint16_t kernelWidth, uint16_t kernelHeight,
	                 void* image,  uint32_t imageWidth,  uint32_t imageHeight,
	                 void* res)
{
	auto imageData  = static_cast<RGBPixel*>(image);
	auto result     = static_cast<RGBPixel*>(res);
	auto kernelData = static_cast<float*>(kernel);

	uint32_t block  = blockIdx.x;
	uint32_t thread = threadIdx.x;

	// a & b loops go through final image
	for (uint32_t a = block; a < imageHeight; a += CUDA_BLOCK_COUNT) // y
	{
		for (uint32_t b = thread; b < imageWidth; b+= CUDA_THREAD_COUNT) // x
		{
			float red = 0;
			float green = 0;
			float blue = 0;

			for (int c = 0; c < kernelHeight; c++) // y
			{
				for (int d = 0; d < kernelWidth; d++) // x
				{
					auto y = c - kernelHeight / 2;
					y = (kernelHeight + a + y) % imageHeight;

					auto x = d - kernelWidth / 2;
					x = (kernelWidth + b + x) % imageWidth;
				
					auto kernelVal = kernelData[c * kernelWidth + d];
					auto imageVal = imageData[y * imageWidth + x];
					if(kernelVal > 0)

					red   += kernelVal * imageVal.red;
					green += kernelVal * imageVal.green;
					blue  += kernelVal * imageVal.blue;
				}
			}

			red   = red   > 0 ? (red   > 255 ? 255 : red)   : 0;
			green = green > 0 ? (green > 255 ? 255 : green) : 0;
			blue  = blue  > 0 ? (blue  > 255 ? 255 : blue)  : 0;

			result[a * imageWidth + b].red   = static_cast<uint8_t>(red);
			result[a * imageWidth + b].green = static_cast<uint8_t>(green);
			result[a * imageWidth + b].blue  = static_cast<uint8_t>(blue);
		}
	}
}

void* CopyToGRAM(uint32_t size, void* data) noexcept
{
	void* gpuData = nullptr;
	hipError_t error;

	// Alocate GPU memory
	error = hipMalloc(&gpuData, size);
	if (error != hipSuccess)
	{
		std::cerr << "Error: Unable to allocate GRAM of size: " << size << std::endl;
		hipFree(gpuData);
		return nullptr;
	}
	hipDeviceSynchronize();

	// Copy Data to GPU memory
	error = hipMemcpy(gpuData, data, size, hipMemcpyHostToDevice);
	if (error != hipSuccess)
	{
		std::cerr << "Error: Unable to copy data to GRAM of size: " << size << std::endl;
		hipFree(gpuData);
		return nullptr;
	}
	hipDeviceSynchronize();
	return gpuData;
}

void* CopyToGRAM(const Image& data) noexcept
{
	auto size = data.GetWidth() * data.GetHeight();

	std::unique_ptr<RGBPixel[]> imageData = std::unique_ptr<RGBPixel[]>(new RGBPixel[size]);

	// Prepare data
	uint64_t location = 0;
	for (uint32_t i = 0; i < data.GetHeight(); i++)
	{
		for (uint32_t j = 0; j < data.GetWidth(); j++)
		{
			auto pixel = data.GetPixel(j, i).ToRGB();
			imageData[location].red   = pixel.red;
			imageData[location].green = pixel.green;
			imageData[location].blue  = pixel.blue;
			location++;
		}
	}
	// Copy to GRAM
	return CopyToGRAM(static_cast<uint32_t>(size * 3), imageData.get());
}

void* CopyToGRAM(const Kernel& data) noexcept
{
	auto size = data.GetWidth() * data.GetHeight();

	std::unique_ptr<float[]> kernelData  = std::unique_ptr<float[]>(new float[size]);
	
	uint32_t location = 0;
	for (uint16_t i = 0; i < data.GetHeight(); i++)
	{
		for (uint16_t j = 0; j < data.GetWidth(); j++)
		{
			kernelData[location++] = data.Get(j, i);
		}
	}
	// Copy to GRAM
	return CopyToGRAM(static_cast<uint32_t>(size * sizeof(float)), kernelData.get());
}

std::shared_ptr<Image> Filter::ApplyFilter()
{
	bool error = false;
	std::shared_ptr<Image> result = std::shared_ptr<Image>(new Image(this->image->GetWidth(), this->image->GetHeight()));

	// Copy data to GPU memory
	void* cudaKernel = CopyToGRAM(this->kernel);
	void* cudaImage  = CopyToGRAM(*this->image.get());
	void* cudaResult = nullptr;

	hipMalloc(&cudaResult, this->image->GetHeight() * this->image->GetWidth() * 3);
	hipDeviceSynchronize();

	if ((cudaKernel != nullptr) && (cudaImage != nullptr) && (cudaResult!=nullptr))
	{
		ApplyFilterCuda<<<CUDA_BLOCK_COUNT, CUDA_THREAD_COUNT >>>
			(cudaKernel, this->kernel.GetWidth(), this->kernel.GetHeight(),
			  cudaImage, this->image->GetWidth(), this->image->GetHeight(),
			 cudaResult);
		hipDeviceSynchronize();

		// Copy result image back from gram
		auto imageSize = this->image->GetWidth() * this->image->GetHeight();
		std::unique_ptr<RGBPixel[]> imageData =
			std::unique_ptr<RGBPixel[]>(new RGBPixel[imageSize]);
		hipMemcpy(imageData.get(), cudaResult, imageSize * 3, hipMemcpyDeviceToHost);
		hipDeviceSynchronize();

		// Postprocess data
		uint64_t loc = 0;
		for (uint32_t i = 0; i < this->image->GetHeight(); i++)
		{
			for (uint32_t j = 0; j < this->image->GetWidth(); j++)
			{
				// Set Pixel
				Pixel pixel = { imageData[loc++] };
				result->SetPixel(j, i, pixel);
			}
		}
	}
	else
	{
		error = true;
	}

	// Cleanup
	if (cudaKernel != nullptr)
	{
		hipFree(cudaKernel);
	}

	if (cudaImage != nullptr)
	{
		hipFree(cudaImage);
	}

	if (cudaResult != nullptr)
	{
		hipFree(cudaResult);
	}

	if (error)
	{
		throw RuntimeException("Error: Unable to allocate data on GPU!");
	}

	return result;
}

#endif /* USE_CUDA */